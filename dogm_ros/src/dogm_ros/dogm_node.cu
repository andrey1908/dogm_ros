#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael Kösel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "dogm_ros/dogm_node.h"
#include "dogm_ros/dogm_ros.h"

#include <dogm/dogm.h>
#include <dogm/dogm_types.h>

#include <dogm_msgs/DynamicOccupancyGrid.h>

#include <tf2_eigen/tf2_eigen.h>
#include <geometry_msgs/TransformStamped.h>
#include <geometry_msgs/Transform.h>

#include <Eigen/Dense>
#include <Eigen/Geometry>

#include <opencv2/opencv.hpp>
#include <opencv2/core/eigen.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudawarping.hpp>

#include "time_measurer.h"

namespace dogm_ros
{

__global__ void setUnknownAsFree(cv::cuda::PtrStepSz<signed char> occupancy_grid);
__global__ void fillMeasurementGrid(dogm::MeasurementCell* __restrict__ measurement_grid, const cv::cuda::PtrStepSzi source,
									float occupancy_threshold);

DOGMRos::DOGMRos(ros::NodeHandle nh, ros::NodeHandle private_nh) 
	: nh_(nh), private_nh_(private_nh), tf_buffer_(), tf_listener_(tf_buffer_), is_first_measurement_(true)
{
	private_nh_.param("map/size", params_.size, 50.0f);
	private_nh_.param("map/resolution", params_.resolution, 0.2f);
	private_nh_.param("particles/particle_count", params_.particle_count, 3 * static_cast<int>(1e6));
	private_nh_.param("particles/new_born_particle_count", params_.new_born_particle_count, 3 * static_cast<int>(1e5));
	private_nh_.param("particles/persistence_probability", params_.persistence_prob, 0.99f);
	private_nh_.param("particles/process_noise_position", params_.stddev_process_noise_position, 0.1f);
	private_nh_.param("particles/process_noise_velocity", params_.stddev_process_noise_velocity, 1.0f);
	private_nh_.param("particles/birth_probability", params_.birth_prob, 0.02f);
	private_nh_.param("particles/velocity_persistent", params_.stddev_velocity, 30.0f);
	private_nh_.param("particles/velocity_birth", params_.init_max_velocity, 30.0f);

	private_nh_.param("frame_id", frame_id_, std::string("base_link"));

	private_nh_.param("opencv_visualization", opencv_visualization_, false);
	private_nh_.param("vis_occupancy_threshold", vis_occupancy_threshold_, 0.6f);
	private_nh_.param("vis_mahalanobis_distance", vis_mahalanobis_distance_, 6.0f);
	private_nh_.param("vis_image_size", vis_image_size_, int(400));

	dogm_map_.reset(new dogm::DOGM(params_));
	CHECK_ERROR(hipMalloc(&measurement_grid_, dogm_map_->grid_cell_count * sizeof(dogm::MeasurementCell)));

	subscriber_ = nh_.subscribe("static_map", 1, &DOGMRos::process, this);
	publisher_ = nh_.advertise<dogm_msgs::DynamicOccupancyGrid>("dynamic_map", 1);
}

DOGMRos::~DOGMRos()
{
	CHECK_ERROR(hipFree(measurement_grid_));
}

void DOGMRos::process(const nav_msgs::OccupancyGrid::ConstPtr& occupancy_grid)
{
	MEASURE_TIME_FROM_HERE(OccupancyGrid2MeasurementGrid);
	occupancyGridToMeasurementGrid(occupancy_grid);
	STOP_TIME_MEASUREMENT(OccupancyGrid2MeasurementGrid);
	
	MEASURE_TIME_FROM_HERE(UpdateDynamicMap);
	ros::Time time_stamp = occupancy_grid->header.stamp;
	if (!is_first_measurement_)
	{
		float dt = (time_stamp - last_time_stamp_).toSec();
		dogm_map_->updateGrid(measurement_grid_, new_x_, new_y_, 0, dt);
	}
	else
	{
		dogm_map_->updateGrid(measurement_grid_, new_x_, new_y_, 0, 0);
		is_first_measurement_ = false;
	}
	last_time_stamp_ = time_stamp;
	STOP_TIME_MEASUREMENT(UpdateDynamicMap);
	
	MEASURE_TIME_FROM_HERE(DynamicMap2ROSMessage);
	dogm_msgs::DynamicOccupancyGrid message;
    dogm_ros::DOGMRosConverter::toDOGMMessage(*dogm_map_, message, occupancy_grid->header.frame_id);
	STOP_TIME_MEASUREMENT(DynamicMap2ROSMessage);
    
	publisher_.publish(message);

	if (opencv_visualization_)
	{
		MEASURE_TIME_FROM_HERE(Visualization);
		cv::Mat occupancy_image = dogm_map_->getOccupancyImage();
		dogm_map_->drawVelocities(occupancy_image, vis_image_size_, 1., vis_occupancy_threshold_, vis_mahalanobis_distance_);
		cv::namedWindow("occupancy_image", cv::WINDOW_NORMAL);
		cv::imshow("occupancy_image", occupancy_image);
		cv::waitKey(1);
		STOP_TIME_MEASUREMENT(Visualization);
	}
}

void DOGMRos::occupancyGridToMeasurementGrid(const nav_msgs::OccupancyGrid::ConstPtr& occupancy_grid, float occupancy_threshold /* 0.5 */)
{
	geometry_msgs::TransformStamped odom_to_robot =
		tf_buffer_.lookupTransform(occupancy_grid->header.frame_id, frame_id_, occupancy_grid->header.stamp, ros::Duration(0.15));
	cv::Mat odom_to_measurement_grid(cv::Mat::eye(cv::Size(3, 3), CV_32F));
	odom_to_measurement_grid.at<float>(0, 2) = odom_to_robot.transform.translation.x / params_.resolution - dogm_map_->grid_size / 2.;
	odom_to_measurement_grid.at<float>(1, 2) = odom_to_robot.transform.translation.y / params_.resolution - dogm_map_->grid_size / 2.;
	new_x_ = odom_to_measurement_grid.at<float>(0, 2) * params_.resolution;
	new_y_ = odom_to_measurement_grid.at<float>(1, 2) * params_.resolution;

	cv::Mat scale_measurement_grid(cv::Mat::eye(cv::Size(3, 3), CV_32F));
	float scale = occupancy_grid->info.resolution / params_.resolution;
	scale_measurement_grid.at<float>(0, 0) *= scale;
	scale_measurement_grid.at<float>(1, 1) *= scale;

	// strange bug: eigen matrices here should have different types (float or double) with
	// eigen matrices in dogm.cu (dogm repository) in function drawVelocities(), otherwise
	// the program will crash
	Eigen::Quaternionf eigen_odom_to_occupancy_grid_quaternion;
	eigen_odom_to_occupancy_grid_quaternion.x() = occupancy_grid->info.origin.orientation.x;
	eigen_odom_to_occupancy_grid_quaternion.y() = occupancy_grid->info.origin.orientation.y;
	eigen_odom_to_occupancy_grid_quaternion.z() = occupancy_grid->info.origin.orientation.z;
	eigen_odom_to_occupancy_grid_quaternion.w() = occupancy_grid->info.origin.orientation.w;
	Eigen::Matrix3f eigen_odom_to_occupancy_grid_rotation = eigen_odom_to_occupancy_grid_quaternion.normalized().toRotationMatrix();
	Eigen::Matrix2f eigen_odom_to_occupancy_grid_rotation_2d = eigen_odom_to_occupancy_grid_rotation.block<2, 2>(0, 0);
	cv::Mat odom_to_occupancy_grid(cv::Mat::eye(cv::Size(3, 3), CV_32F));
	cv::eigen2cv(eigen_odom_to_occupancy_grid_rotation_2d, odom_to_occupancy_grid(cv::Range(0, 2), cv::Range(0, 2)));
	odom_to_occupancy_grid.at<float>(0, 2) = occupancy_grid->info.origin.position.x / occupancy_grid->info.resolution;
	odom_to_occupancy_grid.at<float>(1, 2) = occupancy_grid->info.origin.position.y / occupancy_grid->info.resolution;

	cv::Mat measurement_grid_to_occupancy_grid = odom_to_measurement_grid.inv() * scale_measurement_grid * odom_to_occupancy_grid;

	dim3 blocks(1, 1);
	dim3 threads(16, 16);
	std::vector<signed char> occupancy_grid_data(occupancy_grid->data);
	cv::Mat occupancy_grid_host(cv::Size(occupancy_grid->info.width, occupancy_grid->info.height), CV_8S, occupancy_grid_data.data());
	cv::cuda::GpuMat occupancy_grid_device;
	occupancy_grid_device.upload(occupancy_grid_host);
	setUnknownAsFree<<<blocks, threads>>>(occupancy_grid_device);
	occupancy_grid_device.convertTo(occupancy_grid_device, CV_32S);

	cv::Mat measurement_grid;
    cv::cuda::GpuMat measurement_grid_device;
	cv::cuda::warpAffine(occupancy_grid_device, measurement_grid_device, measurement_grid_to_occupancy_grid(cv::Range(0, 2), cv::Range(0, 3)),
		cv::Size(dogm_map_->grid_size, dogm_map_->grid_size), cv::INTER_LINEAR, cv::BORDER_CONSTANT, cv::Scalar(0));
	fillMeasurementGrid<<<blocks, threads>>>(measurement_grid_, measurement_grid_device, occupancy_threshold);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());
}

__global__ void setUnknownAsFree(cv::cuda::PtrStepSz<signed char> occupancy_grid)
{
	int start_row = blockIdx.y * blockDim.y + threadIdx.y;
	int start_col = blockIdx.x * blockDim.x + threadIdx.x;
	int step_row = blockDim.y * gridDim.y;
	int step_col = blockDim.x * gridDim.x;
	for (int row = start_row; row < occupancy_grid.rows; row += step_row)
	{
		for (int col = start_col; col < occupancy_grid.cols; col += step_col)
		{
			if (occupancy_grid(row, col) < 0)
			{
				occupancy_grid(row, col) = 0;
			}
		}
	}
}

__device__ float clip(float x, float min, float max)
{
	assert(min <= max);
	if (x < min) return min;
	if (x > max) return max;
	return x;
}

__global__ void fillMeasurementGrid(dogm::MeasurementCell* __restrict__ measurement_grid, const cv::cuda::PtrStepSzi source,
									float occupancy_threshold)
{
	int start_row = blockIdx.y * blockDim.y + threadIdx.y;
	int start_col = blockIdx.x * blockDim.x + threadIdx.x;
	int step_row = blockDim.y * gridDim.y;
	int step_col = blockDim.x * gridDim.x;
	const float eps = 0.0001f;
	for (int row = start_row; row < source.rows; row += step_row)
	{
		for (int col = start_col; col < source.cols; col += step_col)
		{
			int index = col + row * source.cols;
			float occ = source(row, col) / 100.f;
			if (occ < occupancy_threshold)
			{
				measurement_grid[index].free_mass = clip(1 - occ, eps, 1 - eps);
				measurement_grid[index].occ_mass = eps;
			}
			else
			{
				measurement_grid[index].free_mass = eps;
				measurement_grid[index].occ_mass = clip(occ, eps, 1 - eps);
			}
			measurement_grid[index].likelihood = 1.0f;
			measurement_grid[index].p_A = 1.0f;
		}
	}
}

} // namespace dogm_ros
